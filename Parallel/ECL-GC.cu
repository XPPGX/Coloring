#include "hip/hip_runtime.h"
/**
 * @author XPPGX
 * @remark
 * 1. Bitmap 的某些operation還沒搞定            (pend)
*/

#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#endif

extern "C"{
    #ifndef cCSR
    #define cCSR
    #include "../Lib/CSR/CSR.h"
    #endif

    #ifndef tTime
    #define tTime
    #include "../Lib/tTime/tTime.h"
    #endif
}

#include <hip/hip_runtime.h>

//define
#define _DEBUG_

//用於紀錄CSRInfo
struct CSRInfo{
    int startAtZero;
    int nodeSize;
    int startNodeID;
    int endNodeID;
    int maxDegree;
    int nodeBitmapIntNum;
};

//用於紀錄DAG的offset
struct NodeInfo{
    //For the offset of DAG_E
    int nodeStartOffset;
    int nodeEndOffset;
    
    //For the bitmap        
    int possColorBitLength;     //[variable]    紀錄bitmap長度
    int bitmapStartOffset;      //[variable]    紀錄在bitmap中的offset，代表從bitmap中的哪一格開始
    int bestColor;              //[variable]    紀錄當前最好的顏色
    int worstColor;             //[variable]    紀錄當前最差的顏色    
};

int reAssignNodeSize(CSR* _csr);
CSRInfo getCsrInfo(CSR* _csr, int _nodeSize);
void checkDevice();

void ECL_GC_Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                    int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                    int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                    struct CSRInfo* _deviceCsrInfo, dim3 _block, dim3 _grid);

__device__ void BitmapSetOperation(int nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo);

__global__ void Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                        int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                        int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                        struct CSRInfo* _deviceCsrInfo);

__global__ void Worklist_DAG_Init(int* _cudaWorkList,   int* _cudaWorkListLength,    int* _cudaCsrDegree,
                                  int* _cudaCsrV,       int* _cudaCsrE,             struct NodeInfo* _cudaNodeInfo,
                                  int* _cudaDAG_E,      unsigned int* _cudaNodeBitmap,  struct CSRInfo* _deviceCsrInfo);

__device__ void bitmapRemoveOnebit( int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    int targetBitLocation, struct CSRInfo* _deviceCsrInfo);

__device__ int bitmapGetBestColor(  int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo);

__device__ int bitmapGetWorstColor( int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo);

__global__ void HighDegreeColoring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo** _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo);

void ECL_GC_HighDegree_Vertex_Coloring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E,
                                        struct NodeInfo* _cudaNodeInfo, unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo,
                                        dim3 _block, dim3 _grid);

__global__ void ECL_GC_InitSub( int _nodeID,            int* _cudaCsrV,     int* _cudaCsrE,
                                int* _cudaCsrDegree,    int* _cudaDAG_E,    int* _subKernelFlag,
                                int* _cudaDAG_nodeEndOffset, struct NodeInfo* _cudaDAG_E_Offset);

int main(int argc, char* argv[]){
    char* datasetPath   = argv[1];
    
    Graph* adjlist = buildGraph(datasetPath);
    CSR* csr = createCSR(adjlist);

    #ifdef _DEBUG_
    // showCSR(csr);
    #endif
    int nodeSize                = reAssignNodeSize(csr);        //取得真正的node數
    CSRInfo hostCsrInfo         = getCsrInfo(csr, nodeSize);    //取得Csr的一些資訊(可擴充)
    printf("MaxDegree = %d, node int element num = %d\n", csr->maxDegree, (csr->maxDegree + 32) / 32);
    int colorBitmapIntElementNum   = csr->csrVSize * ((csr->maxDegree + 32) / 32); //紀錄CSR形式的colorMap要用多少個Unsigned Int組成
    //因為每個node的Bitmap長度都是((csr->maxDegree+32) / 32)，以Dblp為例，每個node都有大約11個int的長度。大概300多bit。
    checkDevice();

#pragma region threadlayout
    dim3 block(32,1);
    dim3 grid((csr->csrVSize + block.x - 1) / block.x, 1);
#pragma endregion

#pragma region copyData
    //Define DevicePointer
    int* cudaCsrV;                              //[Array]   以 CSR 的方式紀錄 nodes
    int* cudaCsrE;                              //[Array]   以 CSR 的方式紀錄 edges
    int* cudaCsrDegree;                         //[Array]   紀錄每個 node 的 degree
    int* cudaWorkList;                          //[Array]   紀錄 degree > 32 的node
    int* cudaColorArr;                          //[Array]   紀錄每個 node 的 color     

    int* cudaWorkListNowIndex;                  //[Variable]紀錄在kernel中的 WorkList 的 Index，用於Atomic operation 將 node 放入 WorkList
    struct CSRInfo* deviceCsrInfo;              //[Variable]紀錄 CSR 的資訊如，從0開始、node數量 
    int* cudaDAG_E;                             //[Array]   以CSR的方式紀錄每個node的鄰居，有哪些是比自己更優先的，「offset一開始照舊用csr->CsrV且如果碰到-1則停下，後續則用cudaDAG_V_offset」
    struct NodeInfo* cudaNodeInfo;              //[Array]   紀錄每個node，自己DAG的start offset與end offset，以及當前最好的顏色、可使用的顏色長度。
    unsigned int* cudaNodeBitmap;               //[Array]   紀錄每個node，自己的Bitmap。
    //Malloc device memory space for DevicePointer
    hipMalloc((void**)&cudaCsrV, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaCsrE, sizeof(int) * csr->csrESize);
    hipMalloc((void**)&cudaCsrDegree, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&deviceCsrInfo, sizeof(CSRInfo));
    hipMalloc((void**)&cudaWorkList, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaColorArr, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaWorkListNowIndex, sizeof(int));
    hipMalloc((void**)&cudaDAG_E, sizeof(int) * csr->csrESize);
    hipMalloc((void**)&cudaNodeInfo, sizeof(NodeInfo) * csr->csrVSize);
    hipMalloc((void**)&cudaNodeBitmap, sizeof(unsigned int) * colorBitmapIntElementNum);
    //Copy data from host to device
    printf("[Execution][Copy Data : Host To Device]...\n");
    hipMemcpy(cudaCsrV, csr->csrV, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaCsrE, csr->csrE, sizeof(int) * csr->csrESize, hipMemcpyHostToDevice);
    hipMemcpy(cudaCsrDegree, csr->csrNodesDegree, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceCsrInfo, &hostCsrInfo, sizeof(CSRInfo), hipMemcpyHostToDevice);
    hipMemset(cudaWorkList, 0, sizeof(int) * csr->csrVSize);
    hipMemset(cudaColorArr, -1, sizeof(int) * csr->csrVSize);
    hipMemset(cudaWorkListNowIndex, 0, sizeof(int));
    hipMemset(cudaDAG_E, -1, sizeof(int)* csr->csrESize);  //cudaDAG_E如果有cell = -1，代表那格是空的
    hipMemset(cudaNodeBitmap, 0, sizeof(unsigned int) * colorBitmapIntElementNum); //0代表不能用的，1代表可以用
    printf("[Finish][Copy Data : Host To Device]~\n");
#pragma endregion

#pragma region Algo
    ECL_GC_Init(cudaCsrV, cudaCsrE, cudaWorkList, cudaWorkListNowIndex, cudaCsrDegree, cudaColorArr,
                cudaDAG_E, cudaNodeBitmap, cudaNodeInfo, deviceCsrInfo, block, grid);

    ECL_GC_HighDegree_Vertex_Coloring(  cudaWorkList, cudaWorkListNowIndex, cudaColorArr,
                                        cudaDAG_E, cudaNodeInfo, cudaNodeBitmap, deviceCsrInfo,
                                        block, grid);
#pragma endregion //Algo

    int* hostCsrDegree = (int*)malloc(sizeof(int) * csr->csrVSize);
    hipMemcpy(hostCsrDegree, cudaCsrDegree, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    #ifdef _DEBUG_
    for(int i = hostCsrInfo.startNodeID ; i <= hostCsrInfo.endNodeID ; i ++){
        // printf("Degree[%d] = {host = %d, device = %d}\n", i, csr->csrNodesDegree[i], hostCsrDegree[i]);
        if(csr->csrNodesDegree[i] != hostCsrDegree[i]){
            // printf("node[i] degree record wrong!!\n");
        }
    }
    #endif

    int* hostWorkList                   = (int*)malloc(sizeof(int) * csr->csrVSize);
    int* hostWorkListNowIndex           = (int*)malloc(sizeof(int));
    int* hostDAG_E                      = (int*)malloc(sizeof(int) * csr->csrESize);
    NodeInfo* hostNodeInfo              = (NodeInfo*)malloc(sizeof(NodeInfo) * csr->csrVSize);
    unsigned int* hostNodeBitmap        = (unsigned int*)malloc(sizeof(unsigned int) * colorBitmapIntElementNum);
    int* hostColorArr                   = (int*)malloc(sizeof(int) * csr->csrVSize);

    printf("[Execution][Copy Data : Device To Host]...\n");
    hipMemcpy(hostWorkList, cudaWorkList, sizeof(int) * nodeSize, hipMemcpyDeviceToHost);
    hipMemcpy(hostWorkListNowIndex, cudaWorkListNowIndex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hostDAG_E, cudaDAG_E, sizeof(int) * csr->csrESize, hipMemcpyDeviceToHost);
    hipMemcpy(hostNodeInfo, cudaNodeInfo, sizeof(NodeInfo) * csr->csrVSize, hipMemcpyDeviceToHost);
    hipMemcpy(hostNodeBitmap, cudaNodeBitmap, sizeof(unsigned int) * colorBitmapIntElementNum, hipMemcpyDeviceToHost);
    hipMemcpy(hostColorArr, cudaColorArr, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    printf("[Finish][Copy Data : Device To Host]~\n");
    #ifdef _DEBUG_
    int count = 0;
    printf("=============In Host==============\n");
    // printf("[ColorArr Checking] : \n");
    for(int i = hostCsrInfo.startNodeID ; i <= hostCsrInfo.endNodeID ; i ++){
        // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", i, hostNodeInfo[i].nodeStartOffset, hostNodeInfo[i].nodeEndOffset);
        if(csr->csrNodesDegree[i] >= 32){
            count ++;
        }
        // printf("node[%d].color = %d\n", i, hostColorArr[i]);
        // printf("%d\n", i);
    }
    //亂數
    time_t t;
    srand(time(&t));
    int arbitraryNode = random() % nodeSize + 1;
    // int arbitraryNode = 104410;
    // int arbitraryNode = 96457; //LiveJournal中，bitmap用量較多的一個node
    // int arbitraryNode = 3984674;
    // for(int nodeIter = hostCsrInfo.startNodeID ; nodeIter <= hostCsrInfo.endNodeID ; nodeIter ++){
    //     if(hostNodeInfo[nodeIter].possColorBitLength == 34){
    //         arbitraryNode = nodeIter;
    //     }
    // }

    //hostDAG_E有被建立好!
    printf("[DAG_E  Checking] : ");
    printf("node[%d] = {", arbitraryNode);
    for(int offset = hostNodeInfo[arbitraryNode].nodeStartOffset ; offset < hostNodeInfo[arbitraryNode].nodeEndOffset ; offset ++){
        printf("%d, ", hostDAG_E[offset]);
    }
    printf("}\n");
    printf("[Bitmap Checking] : node[%d].length = %d", arbitraryNode, hostNodeInfo[arbitraryNode].possColorBitLength);
    if(hostNodeInfo[arbitraryNode].possColorBitLength == 0)
        printf(", degree = %d\n", csr->csrNodesDegree[arbitraryNode]);
    else
        printf("\n");
    printf("node[%d] = {", arbitraryNode);
    for(int intIter = 0 ; intIter < ((csr->maxDegree + 32) / 32) ; intIter ++){
        printf("%u, ", hostNodeBitmap[hostNodeInfo[arbitraryNode].bitmapStartOffset + intIter]);
    }
    printf("}\n");
    printf("\nWorkListNowIndex = %u, HostCount = %d\n", *hostWorkListNowIndex, count);
    #endif
    
    // scanf("%d\n", &stdout);
    printf("\nFF\n");

}


int reAssignNodeSize(CSR* _csr){
    int nodeSize = 0;
    if(_csr->startAtZero){
        nodeSize = _csr->csrVSize - 1;
        // printf("Start at 0, nodeSize = %d\n", nodeSize);
    }
    else{
        nodeSize = _csr->csrVSize - 2;
        // printf("Start at 1, nodeSize = %d\n", nodeSize);
    }
    return nodeSize;
}

CSRInfo getCsrInfo(CSR* _csr, int _nodeSize){
    CSRInfo hostCsrInfo;
    hostCsrInfo.nodeSize            = _nodeSize;                            //紀錄總共有幾個Node
    hostCsrInfo.startAtZero         = _csr->startAtZero;
    hostCsrInfo.startNodeID         = !hostCsrInfo.startAtZero;             //紀錄起始的NodeID
    hostCsrInfo.endNodeID           = _nodeSize - hostCsrInfo.startAtZero; //紀錄結束的NodeID
    hostCsrInfo.maxDegree           = _csr->maxDegree;
    hostCsrInfo.nodeBitmapIntNum    = (_csr->maxDegree + 32) / 32;
    printf("hostCsrInfo = {startNodeID = %d, endNodeID = %d, nodeSize = %d, maxDegree = %d}\n", hostCsrInfo.startNodeID, hostCsrInfo.endNodeID, hostCsrInfo.nodeSize, hostCsrInfo.maxDegree);
    return hostCsrInfo;
}

void checkDevice(){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("device %d : %s\n", dev, deviceProp.name);
}

#pragma region ECL_GC_Init
__device__ void BitmapSetOperation(int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo)
{
    int eachNodeIntElementSize  = (_deviceCsrInfo->maxDegree + 32) / 32;
    _cudaNodeInfo[_nodeID].bitmapStartOffset = _nodeID * eachNodeIntElementSize;
    
    int possBitOccupyIntNum     = _cudaNodeInfo[_nodeID].possColorBitLength / 32;
    int possBitRemainLength     = _cudaNodeInfo[_nodeID].possColorBitLength % 32;
    // if(_nodeID == 96457){
    //     printf("node[%d].PossBitOccupyIntNum = %d\n", _nodeID, possBitOccupyIntNum);
    // }
    //把一次超過32bit的1，用for loop assign
    int intIter;
    for(intIter = 0 ; intIter < possBitOccupyIntNum ; intIter ++){
        (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter] = 0xffffffff;
        // if(_nodeID == 96457){
        //     printf("intIter = %d, bitmap = %u\n", intIter, (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter]);
        // }
    }

    //把剩下的bit也assign進去
    unsigned int remainBit = 0;
    for(int i = 0 ; i < possBitRemainLength ; i ++){
        remainBit |= (1 << i);
    }
    (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter] |= remainBit;
    
    
    if(_nodeID == 307071){
        int best = 0;
        int worst = 0;
        best = bitmapGetBestColor(_nodeID, *_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
        worst = bitmapGetWorstColor(_nodeID, *_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
        printf("node[%d] = {bestColor = %d, WorstColor = %d}\n", _nodeID, best, worst);
    }
    // if(_nodeID == 3984674){
    //     bitmapRemoveOnebit(_nodeID, _cudaNodeBitmap, _cudaNodeInfo, 1, _deviceCsrInfo);
    // }
    // if(_nodeID == 96457){ //在LiveJournal中的奇怪case
    //     printf("node[%d] = {bitmapStartOffset = %d, BitLength = %d, Bitmap = %u}\n", _nodeID, _cudaNodeInfo[_nodeID].bitmapStartOffset, _cudaNodeInfo[_nodeID].possColorBitLength, (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter]);
    // }
}

__global__ void Init(int* _cudaCsrV,             int* _cudaCsrE,         int* _cudaWorkList,
                     int* _cudaWorkListNowIndex, int* _cudaCsrDegree,    int* _cudaColorArr,
                     int* _cudaDAG_E,            unsigned int* _cudaNodeBitmap,
                     struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo)
{
                                
    int tid             = threadIdx.x + blockIdx.x * blockDim.x;
    int nodeID          = tid;
    int nodeDegree      = _cudaCsrDegree[nodeID];
    
    if(_deviceCsrInfo->startNodeID <= nodeID && nodeID <= _deviceCsrInfo->endNodeID){
        // _cudaColorArr的初始化已經用hipMemset做了
        // _cudaColorArr[nodeID] = 0;
        
        if(_cudaCsrDegree[nodeID]  >= 32){
            int NowIndex            = atomicAdd(_cudaWorkListNowIndex, 1);
            _cudaWorkList[NowIndex] = nodeID;
            // printf("NowIndex = %d, Node = %d\n", NowIndex, nodeID);
        }
        else{
            int neighborID      = -1;
            int neighborDegree  = -1;
            _cudaNodeInfo[nodeID].nodeStartOffset   = _cudaCsrV[nodeID];
            _cudaNodeInfo[nodeID].nodeEndOffset     = _cudaCsrV[nodeID];
            for(int offsetIter = _cudaCsrV[nodeID] ; offsetIter < _cudaCsrV[nodeID + 1] ; offsetIter ++){
                neighborID      = _cudaCsrE[offsetIter];
                neighborDegree  = _cudaCsrDegree[neighborID];
                if(nodeDegree < neighborDegree){
                    _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                    _cudaNodeInfo[nodeID].nodeEndOffset ++;
                    // printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeEndOffset, neighborID);
                    //用以下方式使用offset
                    /*
                    for(int offset = _cudaNodeInfo[nodeID].nodeStartOffset ; offset < _cudaNodeInfo[nodeID].nodeEndOffset ; offset ++){
                        //do something
                    }
                    */
                }
                else if(nodeDegree == neighborDegree){ //Breaktie by nodeID and neighborID
                    if(nodeID > neighborID){
                        _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                        _cudaNodeInfo[nodeID].nodeEndOffset ++;
                    }
                }
            }
            #ifdef _DEBUG_
            // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaNodeInfo[nodeID].nodeEndOffset);
            #endif
            _cudaNodeInfo[nodeID].possColorBitLength    = _cudaNodeInfo[nodeID].nodeEndOffset - _cudaNodeInfo[nodeID].nodeStartOffset + 1;
            _cudaNodeInfo[nodeID].bestColor             = 0;
            _cudaNodeInfo[nodeID].worstColor            = _cudaNodeInfo[nodeID].possColorBitLength;
            BitmapSetOperation(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
            //初始化_cudaDAG_E已在hipMemcpy完成
            //Call ECL_GC_InitSub的時候，設置1個block = 32個thread，1個grid = 1個block就可以了
            // ECL_GC_InitSub<<<1, 32>>>(nodeID, _cudaCsrV, _cudaCsrE, _cudaCsrDegree, _cudaDAG_E, &subKernelFlag, _cudaDAG_nodeEndOffset, _cudaNodeInfo);
            // //這裡卡不住
            // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaDAG_nodeEndOffset[nodeID]);
        }
    }
}

__global__ void Worklist_DAG_Init(int* _cudaWorkList,   int* _cudaWorkListLength,    int* _cudaCsrDegree,
                                  int* _cudaCsrV,       int* _cudaCsrE,             struct NodeInfo* _cudaNodeInfo,
                                  int* _cudaDAG_E,      unsigned int* _cudaNodeBitmap,  struct CSRInfo* _deviceCsrInfo)
{   
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < *_cudaWorkListLength){
        int nodeID          = _cudaWorkList[tid];
        int nodeDegree      = _cudaCsrDegree[nodeID];
        
        int neighborID      = -1;
        int neighborDegree  = -1;
        
        _cudaNodeInfo[nodeID].nodeStartOffset   = _cudaCsrV[nodeID];
        _cudaNodeInfo[nodeID].nodeEndOffset     = _cudaCsrV[nodeID];

        for(int offsetIter = _cudaCsrV[nodeID] ; offsetIter < _cudaCsrV[nodeID + 1] ; offsetIter ++){
            neighborID      = _cudaCsrE[offsetIter];
            neighborDegree  = _cudaCsrDegree[neighborID];

            if(nodeDegree < neighborDegree){
                _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                _cudaNodeInfo[nodeID].nodeEndOffset ++;
            }
            else if(nodeDegree == neighborDegree){
                if(nodeID > neighborID){
                    _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                    _cudaNodeInfo[nodeID].nodeEndOffset ++;
                }
            }
        }
        _cudaNodeInfo[nodeID].possColorBitLength    = _cudaNodeInfo[nodeID].nodeEndOffset - _cudaNodeInfo[nodeID].nodeStartOffset + 1;
        _cudaNodeInfo[nodeID].bestColor             = 0;
        _cudaNodeInfo[nodeID].worstColor            = _cudaNodeInfo[nodeID].possColorBitLength;
        BitmapSetOperation(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
    }
}

void ECL_GC_Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                    int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                    int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                    struct CSRInfo* _deviceCsrInfo, dim3 _block, dim3 _grid)
{
    printf("=============In Device============\n");
    printf("[Execution][ECL_GC_Init]...\n");

    printf("\t[Execution][Step1]Init...\n");
    Init<<<_grid, _block>>>(  _cudaCsrV,   _cudaCsrE,   _cudaWorkList,
                            _cudaWorkListNowIndex,   _cudaCsrDegree, _cudaColorArr,
                            _cudaDAG_E,  _cudaNodeBitmap, _cudaNodeInfo,   _deviceCsrInfo);
    hipDeviceSynchronize();
    printf("\t[Finish][Step1]Init~\n");

    printf("\t[Execution][Step2]Worklist_DAG_Init...\n");
    Worklist_DAG_Init<<<_grid, _block>>>( _cudaWorkList,   _cudaWorkListNowIndex,   _cudaCsrDegree,  _cudaCsrV,   _cudaCsrE, 
                                        _cudaNodeInfo,   _cudaDAG_E,  _cudaNodeBitmap, _deviceCsrInfo);
    hipDeviceSynchronize();
    printf("\t[Finish][Step2]Worklist_DAG_Init~\n");

    printf("[Finish][ECL_GC_Init]~\n");
}
#pragma endregion //ECL_GC_Init

__device__ void bitmapRemoveOnebit( int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    int targetBitLocation, struct CSRInfo* _deviceCsrInfo)
{
    int intLocation             = (targetBitLocation - 1) / 32;
    int bitLocationInSingleInt  = (targetBitLocation - 1) % 32;

    unsigned int removeBit      = 1 << bitLocationInSingleInt;
    (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intLocation] &= ~removeBit;
}

__device__ int bitmapGetBestColor(  int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo)
{
    int intIter                 = 0;
    int bestColor               = 0;
    unsigned int unsignValue    = 0;
    for(intIter = 0 ; intIter < _deviceCsrInfo->nodeBitmapIntNum ; intIter ++){
        unsignValue = _cudaNodeBitmap[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter];
        bestColor   = __ffsll(unsignValue); //取得該int的 Least set one 的 index， "1010"會取到 2
        if(bestColor != 0){break;} 
    }
    bestColor      += intIter * 32;
    return bestColor;
}

__device__ int bitmapGetWorstColor( int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo)
{
    int intIter                 = (_cudaNodeInfo[_nodeID].possColorBitLength - 1) / 32;
    int worstColor              = 0;
    unsigned int unsignValue    = 0;
    for(; intIter >= 0 ; intIter --){
        unsignValue     = _cudaNodeBitmap[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter];
        //Get most significant bit 1
        unsignValue    |= (unsignValue >> 1);
        unsignValue    |= (unsignValue >> 2);
        unsignValue    |= (unsignValue >> 4);
        unsignValue    |= (unsignValue >> 8);
        unsignValue    |= (unsignValue >> 16);

        unsignValue     = ((unsignValue + 1) >> 1) | (unsignValue & (1 << 31));
        worstColor      = __ffsll(unsignValue); //取得該int的Least set one的index，但這裡會取到 worst index，因為上面的處理
        if(worstColor != 0){break;}
    }
    worstColor += intIter * 32;
    return worstColor;
}

__global__ void HighDegreeColoring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo** _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo)
{
    int tid         = threadIdx.x + blockIdx.x * blockDim.x;
    int shortcut    = 0;    //紀錄是否可用shortcut，如果shortcut = 1則代表可用shortcut
    int done        = 0;    //紀錄全部的鄰居是否都已塗色
    if(tid == 0){
        printf("tid = %d, cudaWorkListLength = %d\n", tid, *_cudaWorkListLength);
    }
    if(tid < *_cudaWorkListLength){
        int nodeID = _cudaWorkList[tid];
        if(_cudaColorArr[nodeID] == -1){    //如果node還沒塗色
            shortcut    = 1;
            done        = 1;
            int nodeBestColor       = (*_cudaNodeInfo[nodeID]).bestColor;
            printf("node[%d].color = %d\n", nodeID, _cudaColorArr[nodeID]);

            int neighborID          = -1;
            int neighborBestColor   = -1;
            int neighborWorstColor  = -1;
            //Checkout neighbors' color
//BUG in For loop below
            // for(int offsetIter = (*_cudaNodeInfo[nodeID]).nodeStartOffset ; offsetIter < (*_cudaNodeInfo[nodeID]).nodeEndOffset ; offsetIter ++){
            //     neighborID  = _cudaDAG_E[offsetIter];
            //     if(_cudaColorArr[neighborID] != -1){    //如果neighbor已塗色
            //         if(_cudaColorArr[neighborID] == (*_cudaNodeInfo[nodeID]).bestColor){
            //             shortcut = 0;
            //         }
            //         /*remove a bit of possible color of nodeID*/
            //         bitmapRemoveOnebit(nodeID, &_cudaNodeBitmap, *_cudaNodeInfo, _cudaColorArr[neighborID], _deviceCsrInfo);
            //     }
            //     else{
            //         done                = 0;
            //         neighborBestColor   = bitmapGetBestColor(neighborID, _cudaNodeBitmap, *_cudaNodeInfo, _deviceCsrInfo);
            //         neighborWorstColor  = bitmapGetWorstColor(neighborID, _cudaNodeBitmap, *_cudaNodeInfo, _deviceCsrInfo);
            //         if(neighborBestColor <= nodeBestColor && nodeBestColor <= neighborWorstColor){
            //             shortcut        = 0;
            //         }
            //     }
            // }

            // (*_cudaNodeInfo[nodeID]).bestColor = bitmapGetBestColor(nodeID, _cudaNodeBitmap, *_cudaNodeInfo, _deviceCsrInfo);
            // if(done || shortcut){
            //     _cudaColorArr[nodeID] = (*_cudaNodeInfo[nodeID]).bestColor;
            // }
            // else{
            //     *_againFlag = 1;
            // }
        }
    }
}

void ECL_GC_HighDegree_Vertex_Coloring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E,
                                        struct NodeInfo* _cudaNodeInfo, unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo,
                                        dim3 _block, dim3 _grid)
{
    printf("[Execution][ECL_GC_HighDegree_Vertex_Coloring]...\n");
    int* hostAgainFlag = (int*)malloc(sizeof(int));

    int* cudaAgainFlag;
    hipMalloc((void**)&cudaAgainFlag, sizeof(int));
    hipMemset(cudaAgainFlag, 0, sizeof(int));
    printf("faker\n");
    do
    {
        *hostAgainFlag = 0;
        hipMemset(cudaAgainFlag, 0, sizeof(int));

        HighDegreeColoring<<<_grid, _block>>>(  _cudaWorkList, _cudaWorkListLength, _cudaColorArr,
                                                _cudaDAG_E, &_cudaNodeInfo, cudaAgainFlag, 
                                                _cudaNodeBitmap, _deviceCsrInfo);
        hipDeviceSynchronize();

        hipMemcpy(hostAgainFlag, cudaAgainFlag, sizeof(int), hipMemcpyDeviceToHost);

    } while (*hostAgainFlag == 1);

    hipDeviceSynchronize();

    printf("[Finish][ECL_GC_HighDegree_Vertex_Coloring]~\n");
}

__global__ void ECL_GC_InitSub( int _nodeID,            int* _cudaCsrV,     int* _cudaCsrE,
                                int* _cudaCsrDegree,    int* _cudaDAG_E,    int* _subKernelFlag,
                                int* _cudaDAG_nodeEndOffset, struct NodeInfo* _cudaDAG_E_Offset)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // degree 18
    // tid    0~31 只有 0~17有用，tid >= 18的都可以滾，只要tid < 18的做事就好
    if(tid < _cudaCsrDegree[_nodeID]){


        int neighborIndex   = tid;
        int neighborID      = _cudaCsrE[_cudaCsrV[_nodeID] + neighborIndex];
        
        int nodeDegree      = _cudaCsrDegree[_nodeID];
        int neighborDegree  = _cudaCsrDegree[neighborID];
        printf("node[%d] = {neighborIndex = %d, neighborID = %d, nodeDegree = %d, neighborDegree = %d}\n", _nodeID, neighborIndex, neighborID, nodeDegree, neighborDegree);
        
        if(tid == 0){
            //設置node在DAG_E的起始offset
            _cudaDAG_E_Offset[_nodeID].nodeStartOffset  = _cudaCsrV[_nodeID];
            //設置nodeEndOffset成起始的offset，且它之後會一直atomicAdd
            _cudaDAG_nodeEndOffset[_nodeID]             = _cudaCsrV[_nodeID]; 
        }
        // printf("cudaDAG_E_Offset[%d] = {startOffset = %d, EndOffset = %d}\n", _nodeID, _cudaDAG_E_Offset[_nodeID].nodeStartOffset, _cudaDAG_E_Offset[_nodeID].nodeEndOffset);
        //比較優先權(比較nodeDegree跟neighborDegree)
        if(nodeDegree < neighborDegree){
            //atomicAdd有問題，在這裡有+，但是不會去改到原本記憶體的值
            int nowOffset           = atomicAdd(&(_cudaDAG_nodeEndOffset[_nodeID]), 1);
            _cudaDAG_E[nowOffset]   = neighborID;
            printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", _nodeID, nowOffset, neighborID);
        }
        else if(nodeDegree == neighborDegree){ //break tie by nodeID and neighborID
            // if nodeID > neighborID，則代表neighbor會先塗，故neighbor優先於node，所以node的DAG要記錄neighbor
            if(_nodeID > neighborID){
            //atomicAdd有問題，在這裡有+，但是不會去改到原本記憶體的值
                int nowOffset           = atomicAdd(&(_cudaDAG_nodeEndOffset[_nodeID]), 1);
                _cudaDAG_E[nowOffset]   = neighborID;
                printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", _nodeID, nowOffset, neighborID);
            }
            // else if nodeID < neighborID，則代表node會先塗，故node優先於neighbor，所以不用記錄東西
        }
    }

    if(tid == 0){
        printf("============node[%d] Finish\n", _nodeID);
        // *_subKernelFlag = 1;
    }
}
