#include "hip/hip_runtime.h"
/**
 * @author XPPGX
 * @date 2023/11/10
 * @brief This is an implementation of ECL-GC algorithm, which is a parallel graph coloring
*/

#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#endif

extern "C"{
    #ifndef cCSR
    #define cCSR
    #include "../../Lib/CSR/CSR.h"
    #endif

    #ifndef tTime
    #define tTime
    #include "../../Lib/tTime/tTime.h"
    #endif
}

#include <hip/hip_runtime.h>

//define
// #define _NoUse_
#define _DEBUG_
#define RECORD_TimeAndQuality
// #define RECORD_NodeProcessedEachLevel
//用於紀錄CSRInfo
struct CSRInfo{
    int startAtZero;
    int nodeSize;
    int startNodeID;
    int endNodeID;
    int maxDegree;
    int nodeBitmapIntNum;
};

//用於紀錄DAG的offset
struct NodeInfo{
    //For the offset of DAG_E
    int nodeStartOffset;
    int nodeEndOffset;
    
    //For the bitmap        
    int possColorBitLength;     //[variable]    紀錄bitmap長度
    int bitmapStartOffset;      //[variable]    紀錄在bitmap中的offset，代表從bitmap中的哪一格開始
    int bestColor;              //[variable]    紀錄當前最好的顏色
    int worstColor;             //[variable]    紀錄當前最差的顏色    
};

char* getFileNameNoExt(char* _datasetPath);
char* combinePath(char* _str1, char* _str2);
int reAssignNodeSize(CSR* _csr);
CSRInfo getCsrInfo(CSR* _csr, int _nodeSize);
void checkDevice();
void checkOneNodeBitmap(struct NodeInfo* _hostNodeInfo, struct CSR* _csr, unsigned int* _hostNodeBitmap, int* _hostDAG_E, int _nodeID);


void ECL_GC_Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                    int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                    int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                    struct CSRInfo* _deviceCsrInfo, dim3 _block, dim3 _grid);

__device__ void BitmapSetOperation(int nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo);

__global__ void Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                        int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                        int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                        struct CSRInfo* _deviceCsrInfo);

__global__ void Worklist_DAG_Init(int* _cudaWorkList,   int* _cudaWorkListLength,    int* _cudaCsrDegree,
                                  int* _cudaCsrV,       int* _cudaCsrE,             struct NodeInfo* _cudaNodeInfo,
                                  int* _cudaDAG_E,      unsigned int* _cudaNodeBitmap,  struct CSRInfo* _deviceCsrInfo);

__device__ void bitmapRemoveOnebit( int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    int targetBitLocation, struct CSRInfo* _deviceCsrInfo);

__device__ int bitmapGetBestColor(  int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo);

__device__ int bitmapGetWorstColor( int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo);

__global__ void HighDegreeColoring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo, int* _nodeProcessedNum);

void ECL_GC_HighDegree_Vertex_Coloring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E,
                                        struct NodeInfo* _cudaNodeInfo, unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo,
                                        dim3 _block, dim3 _grid);

__device__ void removeNeighborFromDAG_E(int _offsetNow, int _nodeID, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo);

__global__ void LowDegreeColoring(  int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo, int* _nodeProcessedNum);

void ECL_GC_LowDegree_Vertex_Coloring(  int* _cudaColorArr, int* _cudaDAG_E,    struct NodeInfo* _cudaNodeInfo,
                                        unsigned int* _cudaNodeBitmap,          struct CSRInfo* _deviceCsrInfo,  dim3 _block, dim3 _grid);

#ifdef _NoUse_
__global__ void ECL_GC_InitSub( int _nodeID,            int* _cudaCsrV,     int* _cudaCsrE,
                                int* _cudaCsrDegree,    int* _cudaDAG_E,    int* _subKernelFlag,
                                int* _cudaDAG_nodeEndOffset, struct NodeInfo* _cudaDAG_E_Offset);
#endif

int main(int argc, char* argv[]){
    char* datasetPath   = argv[1];
    char* fileName      = getFileNameNoExt(datasetPath);
    // printf("FileName = %s\n", fileName);

    double time1;
    double time2;
    double readDatasetTime                  = 0;
    double dataCopyFromHostToDeviceTime     = 0;
    double ECL_GC_InitTime                  = 0;
    double ECL_GC_HighDegreeColoringTime    = 0;
    double ECL_GC_LowDegreeColoringTime     = 0;
    double dataCopyFromDeviceToHost         = 0;
#pragma region readDataset
    time1 = seconds();

    Graph* adjlist = buildGraph(datasetPath);
    CSR* csr = createCSR(adjlist);

    #ifdef _DEBUG_
    // showCSR(csr);
    #endif
    int nodeSize                = reAssignNodeSize(csr);        //取得真正的node數
    CSRInfo hostCsrInfo         = getCsrInfo(csr, nodeSize);    //取得Csr的一些資訊(可擴充)
    printf("MaxDegree = %d, Each node int element num = %d\n", csr->maxDegree, (csr->maxDegree + 32) / 32);
    int colorBitmapIntElementNum   = csr->csrVSize * ((csr->maxDegree + 32) / 32); //紀錄CSR形式的colorMap要用多少個Unsigned Int組成
    //因為每個node的Bitmap長度都是((csr->maxDegree+32) / 32)，以Dblp為例，每個node都有大約11個int的長度。大概300多bit。
    checkDevice();

    time2 = seconds();
    readDatasetTime = time2 - time1;
#pragma endregion //readDataset

#pragma region threadlayout
    dim3 block(32,1);
    dim3 grid((csr->csrVSize + block.x - 1) / block.x, 1);
#pragma endregion

#pragma region copyData
    time1 = seconds();
    //Define DevicePointer
    int* cudaCsrV;                              //[Array]   以 CSR 的方式紀錄 nodes
    int* cudaCsrE;                              //[Array]   以 CSR 的方式紀錄 edges
    int* cudaCsrDegree;                         //[Array]   紀錄每個 node 的 degree
    int* cudaWorkList;                          //[Array]   紀錄 degree > 32 的node
    int* cudaColorArr;                          //[Array]   紀錄每個 node 的 color     

    int* cudaWorkListNowIndex;                  //[Variable]紀錄在kernel中的 WorkList 的 Index，用於Atomic operation 將 node 放入 WorkList
    struct CSRInfo* deviceCsrInfo;              //[Variable]紀錄 CSR 的資訊如，從0開始、node數量 
    int* cudaDAG_E;                             //[Array]   以CSR的方式紀錄每個node的鄰居，有哪些是比自己更優先的，「offset一開始照舊用csr->CsrV且如果碰到-1則停下，後續則用cudaDAG_V_offset」
    struct NodeInfo* cudaNodeInfo;              //[Array]   紀錄每個node，自己DAG的start offset與end offset，以及當前最好的顏色、可使用的顏色長度。
    unsigned int* cudaNodeBitmap;               //[Array]   紀錄每個node，自己的Bitmap。
    int* cudaNodeDAG_StartOffset_dyn;            //[Array]   在LowDegree塗色時，每個node的StartOffset會變動。
    //Malloc device memory space for DevicePointer
    hipMalloc((void**)&cudaCsrV, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaCsrE, sizeof(int) * csr->csrESize);
    hipMalloc((void**)&cudaCsrDegree, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&deviceCsrInfo, sizeof(CSRInfo));
    hipMalloc((void**)&cudaWorkList, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaColorArr, sizeof(int) * csr->csrVSize);
    hipMalloc((void**)&cudaWorkListNowIndex, sizeof(int));
    hipMalloc((void**)&cudaDAG_E, sizeof(int) * csr->csrESize);
    hipMalloc((void**)&cudaNodeInfo, sizeof(NodeInfo) * csr->csrVSize);
    hipMalloc((void**)&cudaNodeBitmap, sizeof(unsigned int) * colorBitmapIntElementNum);
    hipMalloc((void**)&cudaNodeDAG_StartOffset_dyn, sizeof(int) * csr->csrVSize);
    //Copy data from host to device
    printf("[Execution][Copy Data : Host To Device]...\n");
    hipMemcpy(cudaCsrV, csr->csrV, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaCsrE, csr->csrE, sizeof(int) * csr->csrESize, hipMemcpyHostToDevice);
    hipMemcpy(cudaCsrDegree, csr->csrNodesDegree, sizeof(int) * csr->csrVSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceCsrInfo, &hostCsrInfo, sizeof(CSRInfo), hipMemcpyHostToDevice);
    hipMemset(cudaWorkList, 0, sizeof(int) * csr->csrVSize);
    hipMemset(cudaColorArr, -1, sizeof(int) * csr->csrVSize);
    hipMemset(cudaWorkListNowIndex, 0, sizeof(int));
    hipMemset(cudaDAG_E, -1, sizeof(int)* csr->csrESize);  //cudaDAG_E如果有cell = -1，代表那格是空的
    hipMemset(cudaNodeBitmap, 0, sizeof(unsigned int) * colorBitmapIntElementNum); //0代表不能用的，1代表可以用
    printf("[Finish][Copy Data : Host To Device]~\n");

    time2 = seconds();
    dataCopyFromHostToDeviceTime = time2 - time1;
#pragma endregion

#pragma region Algo
    time1 = seconds();
    ECL_GC_Init(cudaCsrV, cudaCsrE, cudaWorkList, cudaWorkListNowIndex, cudaCsrDegree, cudaColorArr,
                cudaDAG_E, cudaNodeBitmap, cudaNodeInfo, deviceCsrInfo, block, grid);
    time2 = seconds();
    ECL_GC_InitTime = time2 - time1;
    /*********************************/
    time1 = seconds();
    ECL_GC_HighDegree_Vertex_Coloring(  cudaWorkList, cudaWorkListNowIndex, cudaColorArr,
                                        cudaDAG_E, cudaNodeInfo, cudaNodeBitmap, deviceCsrInfo,
                                        block, grid);
    time2 = seconds();
    ECL_GC_HighDegreeColoringTime = time2 - time1;
    /*********************************/
    time1 = seconds();
    ECL_GC_LowDegree_Vertex_Coloring(cudaColorArr, cudaDAG_E, cudaNodeInfo, cudaNodeBitmap, deviceCsrInfo, block, grid);
    time2 = seconds();
    ECL_GC_LowDegreeColoringTime = time2 - time1;
#pragma endregion //Algo

    time1 = seconds();
    printf("[Execution][Copy Data : Device To Host]...\n");

    int* hostColorArr                   = (int*)malloc(sizeof(int) * csr->csrVSize);
    int* hostWorkListNowIndex           = (int*)malloc(sizeof(int));

    hipMemcpy(hostColorArr, cudaColorArr, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    hipMemcpy(hostWorkListNowIndex, cudaWorkListNowIndex, sizeof(int), hipMemcpyDeviceToHost);

    printf("[Finish][Copy Data : Device To Host]~\n\n");
    time2 = seconds();
    dataCopyFromDeviceToHost = time2 - time1;

    //檢查整個graph的顏色是否有衝突
    printf("[Check]Whole graph color confliction...\n");
    int maxColorIndex = 0;
    for(int nodeID = hostCsrInfo.startNodeID ; nodeID <= hostCsrInfo.endNodeID ; nodeID ++){
        int nodeColor = hostColorArr[nodeID];
        if(maxColorIndex < nodeColor){maxColorIndex = nodeColor;}
        int flag = 0;
        if(nodeColor == -1){
            printf("\t[Error][Node not yet colored] : node[%d].color = %d\n", nodeID, nodeColor);
            break;
        }
        for(int neighborIndex = csr->csrV[nodeID] ; neighborIndex < csr->csrV[nodeID + 1] ; neighborIndex ++){
            int neighborID = csr->csrE[neighborIndex];
            int neighborColor = hostColorArr[neighborID];
            if(nodeColor == neighborColor){
                printf("\n\n\t[Error][Neighbors with same color] : node[%d].color = %d, neighbor[%d].color = %d!!!!!!!!!!!!!!!!!\n\n", nodeID, nodeColor, neighborID, neighborColor);
                flag = 1;
                break;
            }
        }
        if(flag == 1){break;}
    }
    printf("[Finish][Copy Data : Device To Host]~\n");
    
    #ifdef RECORD_TimeAndQuality
    printf("====================Result===================\n");
    printf("[Time][ReadDataset]                     : %6f\n", readDatasetTime);
    printf("[Time][DataCopy HostToDevice]           : %6f\n", dataCopyFromHostToDeviceTime);
    printf("[Time][ECL_GC_InitTime]                 : %6f\n", ECL_GC_InitTime);
    printf("[Time][ECL_GC_HighDegreeColoringTime]   : %6f\n", ECL_GC_HighDegreeColoringTime);
    printf("[Time][ECL_GC_LowDegreeColoringTime]    : %6f\n", ECL_GC_LowDegreeColoringTime);
    printf("[Time][dataCopyFromDeviceToHost]        : %6f\n", dataCopyFromDeviceToHost);

    printf("WorkListLength   = %d\n", *hostWorkListNowIndex);
    printf("Total used Color = %d\n", maxColorIndex);


    char* OutputFileName = combinePath(fileName, "_Metrics.csv");
    FILE *fptr = fopen(OutputFileName, "a");
    if(fptr == NULL){
        printf("[Error] OpenFile : TimeAndQuality.csv\n");
        exit(1);
    }
    fprintf(fptr, "%f,%f,%f,%f,%f,%f,%d,%d\n",
            readDatasetTime,    dataCopyFromHostToDeviceTime,   ECL_GC_InitTime,
            ECL_GC_HighDegreeColoringTime,  ECL_GC_LowDegreeColoringTime,   dataCopyFromDeviceToHost,
            *hostWorkListNowIndex,  maxColorIndex);
    fclose(fptr);
    #endif

    // int* hostCsrDegree = (int*)malloc(sizeof(int) * csr->csrVSize);
    // hipMemcpy(hostCsrDegree, cudaCsrDegree, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    // #ifdef _DEBUG_
    // for(int i = hostCsrInfo.startNodeID ; i <= hostCsrInfo.endNodeID ; i ++){
    //     // printf("Degree[%d] = {host = %d, device = %d}\n", i, csr->csrNodesDegree[i], hostCsrDegree[i]);
    //     if(csr->csrNodesDegree[i] != hostCsrDegree[i]){
    //         printf("node[i] degree record wrong!!\n");
    //     }
    // }
    // #endif

    // int* hostWorkList                   = (int*)malloc(sizeof(int) * csr->csrVSize);
    // int* hostWorkListNowIndex           = (int*)malloc(sizeof(int));
    // int* hostDAG_E                      = (int*)malloc(sizeof(int) * csr->csrESize);
    // NodeInfo* hostNodeInfo              = (NodeInfo*)malloc(sizeof(NodeInfo) * csr->csrVSize);
    // unsigned int* hostNodeBitmap        = (unsigned int*)malloc(sizeof(unsigned int) * colorBitmapIntElementNum);
    // int* hostColorArr                   = (int*)malloc(sizeof(int) * csr->csrVSize);

    // printf("[Execution][Copy Data : Device To Host]...\n");
    // hipMemcpy(hostWorkList, cudaWorkList, sizeof(int) * nodeSize, hipMemcpyDeviceToHost);
    // hipMemcpy(hostWorkListNowIndex, cudaWorkListNowIndex, sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(hostDAG_E, cudaDAG_E, sizeof(int) * csr->csrESize, hipMemcpyDeviceToHost);
    // hipMemcpy(hostNodeInfo, cudaNodeInfo, sizeof(NodeInfo) * csr->csrVSize, hipMemcpyDeviceToHost);
    // hipMemcpy(hostNodeBitmap, cudaNodeBitmap, sizeof(unsigned int) * colorBitmapIntElementNum, hipMemcpyDeviceToHost);
    // hipMemcpy(hostColorArr, cudaColorArr, sizeof(int) * csr->csrVSize, hipMemcpyDeviceToHost);
    // printf("[Finish][Copy Data : Device To Host]~\n");
    // #ifdef _DEBUG_
    // int count = 0;
    // printf("=============In Host==============\n");
    // // printf("[ColorArr Checking] : \n");
    // for(int i = hostCsrInfo.startNodeID ; i <= hostCsrInfo.endNodeID ; i ++){
    //     // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", i, hostNodeInfo[i].nodeStartOffset, hostNodeInfo[i].nodeEndOffset);
    //     if(csr->csrNodesDegree[i] >= 32){
    //         count ++;
    //     }
    //     // printf("node[%d].color = %d\n", i, hostColorArr[i]);
    //     // printf("%d\n", i);
    // }

    //檢查WorkList中的顏色是否有衝突
    // printf("[Check]Worklist Color confliction...\n");
    // for(int i = 0 ; i < *hostWorkListNowIndex ; i ++){
    //     int nodeID          = hostWorkList[i];
    //     int nodeColor       = hostColorArr[nodeID];
    //     int flag            = 0;
    //     if(nodeColor == -1){
    //         printf("\t[Error][Node not yet colored] : node[%d].color = %d\n", nodeID, nodeColor);
    //         break;
    //     }
    //     for(int neighborIndex = csr->csrV[nodeID] ; neighborIndex < csr->csrV[nodeID + 1] ; neighborIndex ++){
    //         int neighborID = csr->csrE[neighborIndex];
    //         int neighborColor = hostColorArr[neighborID];
    //         if(nodeColor == neighborColor){
    //             printf("\n\n\t[Error][Neighbors with same color] : node[%d].color = %d, neighbor[%d].color = %d!!!!!!!!!!!!!!!!!\n\n", nodeID, nodeColor, neighborID, neighborColor);
    //             flag = 1;
    //             break;
    //         }
    //     }
    //     if(flag == 1){break;}
    // }

    //檢查整個graph的顏色是否有衝突
    // printf("[Check]Whole graph color confliction...\n");
    // int maxColorIndex = 0;
    // for(int nodeID = hostCsrInfo.startNodeID ; nodeID <= hostCsrInfo.endNodeID ; nodeID ++){
    //     int nodeColor = hostColorArr[nodeID];
    //     if(maxColorIndex < nodeColor){maxColorIndex = nodeColor;}
    //     int flag = 0;
    //     if(nodeColor == -1){
    //         printf("\t[Error][Node not yet colored] : node[%d].color = %d\n", nodeID, nodeColor);
    //         break;
    //     }
    //     for(int neighborIndex = csr->csrV[nodeID] ; neighborIndex < csr->csrV[nodeID + 1] ; neighborIndex ++){
    //         int neighborID = csr->csrE[neighborIndex];
    //         int neighborColor = hostColorArr[neighborID];
    //         if(nodeColor == neighborColor){
    //             printf("\n\n\t[Error][Neighbors with same color] : node[%d].color = %d, neighbor[%d].color = %d!!!!!!!!!!!!!!!!!\n\n", nodeID, nodeColor, neighborID, neighborColor);
    //             flag = 1;
    //             break;
    //         }
    //     }
    //     if(flag == 1){break;}
    // }

    // //亂數
    // time_t t;
    // srand(time(&t));
    // int arbitraryNode = random() % nodeSize + 1;
    // checkOneNodeBitmap(hostNodeInfo, csr, hostNodeBitmap, hostDAG_E, arbitraryNode);
    // int arbitraryNode = 104410;

    //檢查特定node的hostDAG_E是否有被建立好
    // int node1 = 553341;
    // checkOneNodeBitmap(hostNodeInfo, csr, hostNodeBitmap, hostDAG_E, node1);

    // printf("\nWorkListNowIndex = %u, HostCount = %d\n", *hostWorkListNowIndex, count);
    // #endif
}

char* getFileNameNoExt(char* _datasetPath){
    char* fileNameWithoutExt = (char*)malloc(sizeof(char) * 256);

    char* fileNameStart = strrchr(_datasetPath, '/');
    if(fileNameStart != NULL){
        fileNameStart++;
    }
    else{
        fileNameStart = _datasetPath;
    }

    char* dot = strrchr(fileNameStart, '.');
    if(dot != NULL){
        strncpy(fileNameWithoutExt, fileNameStart, dot - fileNameStart);
        fileNameWithoutExt[dot - fileNameStart] = '\0';
        // printf("File without ext : %s\n", fileNameWithoutExt);
    }
    else{
        // printf("File has no ext : %s\n", fileNameStart);
    }
    return fileNameWithoutExt;
}

char* combinePath(char* _str1, char* _str2){
    size_t fullPathLen = strlen(_str1) + strlen(_str2) + 1;
    char* fullPath = (char*)malloc(sizeof(char) * fullPathLen);
    strcpy(fullPath, _str1);
    strcat(fullPath, _str2);
    printf("Full path : %s\n", fullPath);
    return fullPath;
}

int reAssignNodeSize(CSR* _csr){
    int nodeSize = 0;
    if(_csr->startAtZero){
        nodeSize = _csr->csrVSize - 1;
        // printf("Start at 0, nodeSize = %d\n", nodeSize);
    }
    else{
        nodeSize = _csr->csrVSize - 2;
        // printf("Start at 1, nodeSize = %d\n", nodeSize);
    }
    return nodeSize;
}

CSRInfo getCsrInfo(CSR* _csr, int _nodeSize){
    CSRInfo hostCsrInfo;
    hostCsrInfo.nodeSize            = _nodeSize;                            //紀錄總共有幾個Node
    hostCsrInfo.startAtZero         = _csr->startAtZero;
    hostCsrInfo.startNodeID         = !hostCsrInfo.startAtZero;             //紀錄起始的NodeID
    hostCsrInfo.endNodeID           = _nodeSize - hostCsrInfo.startAtZero; //紀錄結束的NodeID
    hostCsrInfo.maxDegree           = _csr->maxDegree;
    hostCsrInfo.nodeBitmapIntNum    = (_csr->maxDegree + 32) / 32;
    printf("hostCsrInfo = {startNodeID = %d, endNodeID = %d, nodeSize = %d, maxDegree = %d}\n", hostCsrInfo.startNodeID, hostCsrInfo.endNodeID, hostCsrInfo.nodeSize, hostCsrInfo.maxDegree);
    return hostCsrInfo;
}

void checkDevice(){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("device %d : %s\n", dev, deviceProp.name);
    hipDeviceReset();
}

void checkOneNodeBitmap(struct NodeInfo* _hostNodeInfo, struct CSR* _csr, unsigned int* _hostNodeBitmap, int* _hostDAG_E, int _nodeID){
    printf("[DAG_E  Checking] : ");
    printf("node[%d] = {", _nodeID);
    for(int offset = _hostNodeInfo[_nodeID].nodeStartOffset ; offset < _hostNodeInfo[_nodeID].nodeEndOffset ; offset ++){
        printf("%d, ", _hostDAG_E[offset]);
    }
    printf("}\n");
    printf("[Bitmap Checking] : node[%d].length = %d", _nodeID, _hostNodeInfo[_nodeID].possColorBitLength);
    if(_hostNodeInfo[_nodeID].possColorBitLength == 0)
        printf(", degree = %d\n", _csr->csrNodesDegree[_nodeID]);
    else
        printf("\n");
    printf("node[%d] = {", _nodeID);
    for(int intIter = 0 ; intIter < ((_csr->maxDegree + 32) / 32) ; intIter ++){
        printf("%x, ", _hostNodeBitmap[_hostNodeInfo[_nodeID].bitmapStartOffset + intIter]);
    }
    printf("}\n\n");
}

#pragma region ECL_GC_Init
__device__ void BitmapSetOperation(int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo)
{
    int eachNodeIntElementSize  = (_deviceCsrInfo->maxDegree + 32) / 32;
    _cudaNodeInfo[_nodeID].bitmapStartOffset = _nodeID * eachNodeIntElementSize;
    
    int possBitOccupyIntNum     = _cudaNodeInfo[_nodeID].possColorBitLength / 32;
    int possBitRemainLength     = _cudaNodeInfo[_nodeID].possColorBitLength % 32;
    // if(_nodeID == 96457){
    //     printf("node[%d].PossBitOccupyIntNum = %d\n", _nodeID, possBitOccupyIntNum);
    // }

    //把一次超過32bit的1，用for loop assign
    int intIter;
    for(intIter = 0 ; intIter < possBitOccupyIntNum ; intIter ++){
        (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter] = 0xffffffff;
        // if(_nodeID == 96457){
        //     printf("intIter = %d, bitmap = %u\n", intIter, (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter]);
        // }
    }

    //把剩下的bit也assign進去
    unsigned int remainBit = 0;
    for(int i = 0 ; i < possBitRemainLength ; i ++){
        remainBit |= (1 << i);
    }
    (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter] |= remainBit;
    
    // if(_nodeID == 3984674){
    //     bitmapRemoveOnebit(_nodeID, _cudaNodeBitmap, _cudaNodeInfo, 1, _deviceCsrInfo);
    // }
    // if(_nodeID == 96457){ //在LiveJournal中的奇怪case
    //     printf("node[%d] = {bitmapStartOffset = %d, BitLength = %d, Bitmap = %u}\n", _nodeID, _cudaNodeInfo[_nodeID].bitmapStartOffset, _cudaNodeInfo[_nodeID].possColorBitLength, (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter]);
    // }
}

__global__ void Init(int* _cudaCsrV,             int* _cudaCsrE,         int* _cudaWorkList,
                     int* _cudaWorkListNowIndex, int* _cudaCsrDegree,    int* _cudaColorArr,
                     int* _cudaDAG_E,            unsigned int* _cudaNodeBitmap,
                     struct NodeInfo* _cudaNodeInfo, struct CSRInfo* _deviceCsrInfo)
{
                                
    int tid             = threadIdx.x + blockIdx.x * blockDim.x;
    int nodeID          = tid;
    int nodeDegree      = _cudaCsrDegree[nodeID];
    
    if(_deviceCsrInfo->startNodeID <= nodeID && nodeID <= _deviceCsrInfo->endNodeID){
        // _cudaColorArr的初始化已經用hipMemset做了
        // _cudaColorArr[nodeID] = 0;
        
        if(_cudaCsrDegree[nodeID]  >= 32){
            int NowIndex            = atomicAdd(_cudaWorkListNowIndex, 1);
            _cudaWorkList[NowIndex] = nodeID;
            // printf("NowIndex = %d, Node = %d\n", NowIndex, nodeID);
        }
        else{
            int neighborID      = -1;
            int neighborDegree  = -1;
            _cudaNodeInfo[nodeID].nodeStartOffset   = _cudaCsrV[nodeID];
            _cudaNodeInfo[nodeID].nodeEndOffset     = _cudaCsrV[nodeID];
            for(int offsetIter = _cudaCsrV[nodeID] ; offsetIter < _cudaCsrV[nodeID + 1] ; offsetIter ++){
                neighborID      = _cudaCsrE[offsetIter];
                neighborDegree  = _cudaCsrDegree[neighborID];
                if(nodeDegree < neighborDegree){
                    _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                    _cudaNodeInfo[nodeID].nodeEndOffset ++;
                    // printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeEndOffset, neighborID);
                    //用以下方式使用offset
                    /*
                    for(int offset = _cudaNodeInfo[nodeID].nodeStartOffset ; offset < _cudaNodeInfo[nodeID].nodeEndOffset ; offset ++){
                        //do something
                    }
                    */
                }
                else if(nodeDegree == neighborDegree){ //Breaktie by nodeID and neighborID
                    if(nodeID > neighborID){
                        _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                        _cudaNodeInfo[nodeID].nodeEndOffset ++;
                    }
                }
            }
            #ifdef _DEBUG_
            // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaNodeInfo[nodeID].nodeEndOffset);
            #endif
            _cudaNodeInfo[nodeID].possColorBitLength    = _cudaNodeInfo[nodeID].nodeEndOffset - _cudaNodeInfo[nodeID].nodeStartOffset + 1;
            _cudaNodeInfo[nodeID].bestColor             = 0;
            _cudaNodeInfo[nodeID].worstColor            = _cudaNodeInfo[nodeID].possColorBitLength;
            BitmapSetOperation(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
            //初始化_cudaDAG_E已在hipMemcpy完成
            // printf("node[%d] = {startOffset = %d, endOffset = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaDAG_nodeEndOffset[nodeID]);
        }
    }
}

__global__ void Worklist_DAG_Init(int* _cudaWorkList,   int* _cudaWorkListLength,    int* _cudaCsrDegree,
                                  int* _cudaCsrV,       int* _cudaCsrE,             struct NodeInfo* _cudaNodeInfo,
                                  int* _cudaDAG_E,      unsigned int* _cudaNodeBitmap,  struct CSRInfo* _deviceCsrInfo)
{   
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < *_cudaWorkListLength){
        int nodeID          = _cudaWorkList[tid];
        int nodeDegree      = _cudaCsrDegree[nodeID];
        
        int neighborID      = -1;
        int neighborDegree  = -1;
        
        _cudaNodeInfo[nodeID].nodeStartOffset   = _cudaCsrV[nodeID];
        _cudaNodeInfo[nodeID].nodeEndOffset     = _cudaCsrV[nodeID];

        for(int offsetIter = _cudaCsrV[nodeID] ; offsetIter < _cudaCsrV[nodeID + 1] ; offsetIter ++){
            neighborID      = _cudaCsrE[offsetIter];
            neighborDegree  = _cudaCsrDegree[neighborID];

            if(nodeDegree < neighborDegree){
                _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                _cudaNodeInfo[nodeID].nodeEndOffset ++;
            }
            else if(nodeDegree == neighborDegree){
                if(nodeID > neighborID){
                    _cudaDAG_E[_cudaNodeInfo[nodeID].nodeEndOffset] = neighborID;
                    _cudaNodeInfo[nodeID].nodeEndOffset ++;
                }
            }
        }
        _cudaNodeInfo[nodeID].possColorBitLength    = _cudaNodeInfo[nodeID].nodeEndOffset - _cudaNodeInfo[nodeID].nodeStartOffset + 1;
        _cudaNodeInfo[nodeID].bestColor             = 0;
        _cudaNodeInfo[nodeID].worstColor            = _cudaNodeInfo[nodeID].possColorBitLength;
        BitmapSetOperation(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
    }
}

void ECL_GC_Init(   int* _cudaCsrV, int* _cudaCsrE, int* _cudaWorkList,
                    int* _cudaWorkListNowIndex, int* _cudaCsrDegree, int* _cudaColorArr,
                    int* _cudaDAG_E, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                    struct CSRInfo* _deviceCsrInfo, dim3 _block, dim3 _grid)
{
    printf("=============In Device============\n");
    printf("[Execution][ECL_GC_Init]...\n");

    printf("\t[Execution][Step1]Init...\n");
    Init<<<_grid, _block>>>(  _cudaCsrV,   _cudaCsrE,   _cudaWorkList,
                            _cudaWorkListNowIndex,   _cudaCsrDegree, _cudaColorArr,
                            _cudaDAG_E,  _cudaNodeBitmap, _cudaNodeInfo,   _deviceCsrInfo);
    hipDeviceSynchronize();

    printf("\t[Finish][Step1]Init~\n");
    printf("\n");

    printf("\t[Execution][Step2]Worklist_DAG_Init...\n");
    Worklist_DAG_Init<<<_grid, _block>>>( _cudaWorkList,   _cudaWorkListNowIndex,   _cudaCsrDegree,  _cudaCsrV,   _cudaCsrE, 
                                        _cudaNodeInfo,   _cudaDAG_E,  _cudaNodeBitmap, _deviceCsrInfo);
    hipDeviceSynchronize();
    printf("\t[Finish][Step2]Worklist_DAG_Init~\n");

    printf("[Finish][ECL_GC_Init]~\n\n");
}
#pragma endregion //ECL_GC_Init

__device__ void bitmapRemoveOnebit( int _nodeID, unsigned int** _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    int targetBitLocation, struct CSRInfo* _deviceCsrInfo)
{
    int intLocation             = (targetBitLocation - 1) / 32;
    int bitLocationInSingleInt  = (targetBitLocation - 1) % 32;

    unsigned int removeBit      = 1 << bitLocationInSingleInt;
    (*_cudaNodeBitmap)[_cudaNodeInfo[_nodeID].bitmapStartOffset + intLocation] &= ~removeBit;
}

__device__ int bitmapGetBestColor(  int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo)
{
    int intIter                 = 0;
    int bestColor               = 0;
    unsigned int unsignValue    = 0;
    for(intIter = 0 ; intIter < _deviceCsrInfo->nodeBitmapIntNum ; intIter ++){
        unsignValue = _cudaNodeBitmap[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter];
        bestColor   = __ffsll(unsignValue); //取得該int的 Least set one 的 index， "1010"會取到 2
        if(bestColor != 0){break;} 
    }
    bestColor      += intIter * 32;
    return bestColor;
}

__device__ int bitmapGetWorstColor( int _nodeID, unsigned int* _cudaNodeBitmap, struct NodeInfo* _cudaNodeInfo,
                                    struct CSRInfo* _deviceCsrInfo)
{
    int intIter                 = (_cudaNodeInfo[_nodeID].possColorBitLength - 1) / 32;
    int worstColor              = 0;
    unsigned int unsignValue    = 0;
    for(; intIter >= 0 ; intIter --){
        unsignValue     = _cudaNodeBitmap[_cudaNodeInfo[_nodeID].bitmapStartOffset + intIter];
        //Get most significant bit 1
        unsignValue    |= (unsignValue >> 1);
        unsignValue    |= (unsignValue >> 2);
        unsignValue    |= (unsignValue >> 4);
        unsignValue    |= (unsignValue >> 8);
        unsignValue    |= (unsignValue >> 16);

        unsignValue     = ((unsignValue + 1) >> 1) | (unsignValue & (1 << 31));
        worstColor      = __ffsll(unsignValue); //取得該int的Least set one的index，但這裡會取到 worst index，因為上面的處理
        if(worstColor != 0){break;}
    }
    worstColor += intIter * 32;
    return worstColor;
}

#pragma region ECL_GC_High_Degree_Node_Coloring

__global__ void HighDegreeColoring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo, int* _nodeProcessedNum)
{
    int tid         = threadIdx.x + blockIdx.x * blockDim.x;
    int shortcut    = 0;    //紀錄是否可用shortcut，如果shortcut = 1則代表可用shortcut
    int done        = 0;    //紀錄全部的鄰居是否都已塗色
    // if(tid == 0){
    //     printf("tid = %d, cudaWorkListLength = %d\n", tid, *_cudaWorkListLength);
    // }
    if(tid < *_cudaWorkListLength){
        int nodeID = _cudaWorkList[tid];
        if(_cudaColorArr[nodeID] == -1){    //如果node還沒塗色
            shortcut    = 1;
            done        = 1;

            int nodeBestColor       = _cudaNodeInfo[nodeID].bestColor;
            // printf("node[%d] = {bestcolor = %d}\n", nodeID, nodeBestColor);

            int neighborID          = -1;
            int neighborBestColor   = -1;
            int neighborWorstColor  = -1;

            //Checkout neighbors' color
            for(int offsetIter = _cudaNodeInfo[nodeID].nodeStartOffset ; offsetIter < _cudaNodeInfo[nodeID].nodeEndOffset ; offsetIter ++){
                neighborID  = _cudaDAG_E[offsetIter];
                if(_cudaColorArr[neighborID] != -1){    //如果neighbor已塗色
                    if(_cudaColorArr[neighborID] == nodeBestColor){
                        shortcut = 0;
                    }
                    //remove a bit of possible color of nodeID
                    bitmapRemoveOnebit(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, _cudaColorArr[neighborID], _deviceCsrInfo);
                    nodeBestColor = bitmapGetBestColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
                }
                else{
                    done                = 0;
                    neighborBestColor   = bitmapGetBestColor(neighborID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
                    neighborWorstColor  = bitmapGetWorstColor(neighborID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);

                    nodeBestColor       = bitmapGetBestColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
                    if((neighborBestColor <= nodeBestColor) && (nodeBestColor <= neighborWorstColor)){
                        shortcut        = 0;
                    }
                }
            }

            // _cudaNodeInfo[nodeID].bestColor = bitmapGetBestColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
            nodeBestColor               = bitmapGetBestColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);

            if(done || shortcut){
                _cudaColorArr[nodeID]   = nodeBestColor;

                #ifdef RECORD_NodeProcessedEachLevel
                int temp = atomicAdd(_nodeProcessedNum, 1);
                #endif
                // _cudaColorArr[nodeID] = _cudaNodeInfo[nodeID].bestColor;
                // if(nodeID == 553341 || nodeID == 1193315){
                //     printf("nodeID = %d, nodeBestColor = %d, cudaNodeInfo.bestColor = %d, neighborID = %d, neighborBestColor = %d, neighborWorstColor = %d\n", nodeID, nodeBestColor, _cudaNodeInfo[nodeID].bestColor, neighborID, neighborBestColor, neighborWorstColor);
                // }
                // printf("node[%d].colored = %d\n", nodeID, _cudaColorArr[nodeID]);
            }
            else{
                *_againFlag = 1;
            }
        }
    }
}

void ECL_GC_HighDegree_Vertex_Coloring( int* _cudaWorkList, int* _cudaWorkListLength, int* _cudaColorArr, int* _cudaDAG_E,
                                        struct NodeInfo* _cudaNodeInfo, unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo,
                                        dim3 _block, dim3 _grid)
{
    printf("[Execution][ECL_GC_HighDegree_Vertex_Coloring]...\n");
    int* hostAgainFlag          = (int*)malloc(sizeof(int));
    int* hostNodeProcessedNum   = (int*)malloc(sizeof(int));

    int* cudaAgainFlag;
    int* cudaNodeProcessedNum;
    hipMalloc((void**)&cudaAgainFlag, sizeof(int));
    hipMalloc((void**)&cudaNodeProcessedNum, sizeof(int));
    hipMemset(cudaNodeProcessedNum, 0, sizeof(int));
    hipMemset(cudaAgainFlag, 0, sizeof(int));

    #ifdef RECORD_NodeProcessedEachLevel
    FILE *fptr = fopen("HD_Nodes_Each_Level.csv", "w");
    if(fptr == NULL){
        printf("[Error] OpenFile : HD_Nodes_Each_Level.csv\n");
        exit(1);
    }
    #endif
    
    int loopCounter             = 0;
    do
    {
        *hostAgainFlag          = 0;
        hipMemset(cudaAgainFlag, 0, sizeof(int));

        #ifdef RECORD_NodeProcessedEachLevel
        *hostNodeProcessedNum   = 0;
        hipMemset(cudaNodeProcessedNum, 0, sizeof(int));
        #endif

        HighDegreeColoring<<<_grid, _block>>>(  _cudaWorkList, _cudaWorkListLength, _cudaColorArr,
                                                _cudaDAG_E, _cudaNodeInfo, cudaAgainFlag, 
                                                _cudaNodeBitmap, _deviceCsrInfo, cudaNodeProcessedNum);
        hipDeviceSynchronize();

        hipMemcpy(hostAgainFlag, cudaAgainFlag, sizeof(int), hipMemcpyDeviceToHost);

        loopCounter ++;

        #ifdef RECORD_NodeProcessedEachLevel
        hipMemcpy(hostNodeProcessedNum, cudaNodeProcessedNum, sizeof(int), hipMemcpyDeviceToHost);
        printf("\tLoop[%d].nodeProcessedNum = %d\n", loopCounter, *hostNodeProcessedNum);
        fprintf(fptr, "%d,%d\n", loopCounter, *hostNodeProcessedNum);
        #endif

    } while (*hostAgainFlag == 1);
    printf("\tLoopCounter = %d\n", loopCounter);
    hipDeviceSynchronize();

    #ifdef RECORD_NodeProcessedEachLevel
    fclose(fptr);
    #endif

    printf("[Finish][ECL_GC_HighDegree_Vertex_Coloring]~\n\n");
}

#pragma endregion //ECL_GC_High_Degree_Node_Coloring

__device__ void removeNeighborFromDAG_E(int _offsetNow, int _nodeID, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo){
    //swap
    int remainNeighbor      = _cudaDAG_E[_cudaNodeInfo[_nodeID].nodeStartOffset];
    _cudaDAG_E[_cudaNodeInfo[_nodeID].nodeStartOffset] = _cudaDAG_E[_offsetNow];
    _cudaDAG_E[_offsetNow]  = remainNeighbor;

    _cudaNodeInfo[_nodeID].nodeStartOffset      ++;
    _cudaNodeInfo[_nodeID].possColorBitLength   --;
}

#pragma region ECL_GC_Low_Degree_Node_Coloring

__global__ void LowDegreeColoring(  int* _cudaColorArr, int* _cudaDAG_E, struct NodeInfo* _cudaNodeInfo,
                                    int* _againFlag,    unsigned int* _cudaNodeBitmap, struct CSRInfo* _deviceCsrInfo, int* _nodeProcessedNum)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(_deviceCsrInfo->startNodeID <= tid && tid <= _deviceCsrInfo->endNodeID){
        int nodeID          = tid;
        int nodeBestColor   = -1;
        if(_cudaColorArr[nodeID] == -1){
            unsigned int Union          = 0x00000000;
            int nodeRemainDAG_Counter   = _cudaNodeInfo[nodeID].possColorBitLength - 1; //若有node x的DAG_E有兩個neignbor，則node x的possColorBitLength = 3
            int nodeWorstColor          = -1;

            int neighborID              = -1;
            int neighborColor           = -1;
            
            for(int offsetIter = _cudaNodeInfo[nodeID].nodeStartOffset ; offsetIter < _cudaNodeInfo[nodeID].nodeEndOffset ; offsetIter ++){
                neighborID  = _cudaDAG_E[offsetIter];
                Union      |= _cudaNodeBitmap[_cudaNodeInfo[neighborID].bitmapStartOffset]; //因為LowDegree node的degree < 32，則node可使用顏色數最多為32，故Union也只需考慮其他neighbor.bitmap的前32個bit就好
                
                
                if(_cudaNodeBitmap[_cudaNodeInfo[nodeID].bitmapStartOffset] & _cudaNodeBitmap[_cudaNodeInfo[neighborID].bitmapStartOffset] == 0x00000000){
                    nodeRemainDAG_Counter --;
                    nodeWorstColor      = bitmapGetWorstColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);

                    // if(nodeID == 3185){
                    //     printf("\tnode[%d].bitmap = %x, neighbor[%d].color = %d\n", nodeID, _cudaNodeBitmap[_cudaNodeInfo[nodeID].bitmapStartOffset], neighborID, _cudaColorArr[neighborID]);
                    // }
                    bitmapRemoveOnebit(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, nodeWorstColor, _deviceCsrInfo);
                    removeNeighborFromDAG_E(offsetIter, nodeID, _cudaDAG_E, _cudaNodeInfo);
                    // if(nodeID == 34970){
                    //     printf("\t[Shortcut 2] : node[%d] = {nodeDAG_offset = %d, neighborID = %d, neighborColor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, neighborID, _cudaColorArr[neighborID]);
                    //     // printf("[Shortcut 2] : node[%d] = {DAG_StartOffset = %d, ColorBitmap = %x, neighborID = %d, neighborColor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaNodeBitmap[_cudaNodeInfo[nodeID].bitmapStartOffset], neighborID, neighborColor);
                    // }
                }
                else if(_cudaColorArr[neighborID] != -1){
                    nodeRemainDAG_Counter --;
                    neighborColor       = _cudaColorArr[neighborID];
                    bitmapRemoveOnebit(nodeID, &_cudaNodeBitmap, _cudaNodeInfo, neighborColor, _deviceCsrInfo);
                    removeNeighborFromDAG_E(offsetIter, nodeID, _cudaDAG_E, _cudaNodeInfo);
                    // if(nodeID == 34970){
                    //     printf("\t[Ordinary] : node[%d] = {nodeDAG_offset = %d, neighborID = %d, neighborColor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, neighborID, _cudaColorArr[neighborID]);
                    //     // printf("[Ordinary] : node[%d] = {DAG_StartOffset = %d, ColorBitmap = %x, neighborID = %d, neighborColor = %d}\n", nodeID, _cudaNodeInfo[nodeID].nodeStartOffset, _cudaNodeBitmap[_cudaNodeInfo[nodeID].bitmapStartOffset], neighborID, neighborColor);
                    // }
                }
            }

            nodeBestColor   = bitmapGetBestColor(nodeID, _cudaNodeBitmap, _cudaNodeInfo, _deviceCsrInfo);
            if((nodeRemainDAG_Counter == 0) || ((Union & (1 << (nodeBestColor - 1)) == 0x00000000))){

                _cudaColorArr[nodeID]   = nodeBestColor;

                #ifdef  RECORD_NodeProcessedEachLevel
                int temp = atomicAdd(_nodeProcessedNum, 1);
                #endif
            }
            else{
                *_againFlag = 1;
            }
        }
    }
}

void ECL_GC_LowDegree_Vertex_Coloring(  int* _cudaColorArr, int* _cudaDAG_E,    struct NodeInfo* _cudaNodeInfo,
                                        unsigned int* _cudaNodeBitmap,          struct CSRInfo* _deviceCsrInfo,  dim3 _block, dim3 _grid)
{
    printf("[Execution][ECL_GC_LowDegree_Vertex_Coloring]...\n");
    int* hostAgainFlag          = (int*)malloc(sizeof(int));
    int* hostNodeProcessedNum   = (int*)malloc(sizeof(int));

    int* cudaAgainFlag;
    int* cudaNodeProcessedNum;
    hipMalloc((void**)&cudaAgainFlag, sizeof(int));
    hipMalloc((void**)&cudaNodeProcessedNum, sizeof(int));
    hipMemset(cudaAgainFlag, 0, sizeof(int));
    hipMemset(cudaNodeProcessedNum, 0, sizeof(int));

    #ifdef RECORD_NodeProcessedEachLevel
    FILE* fptr = fopen("LD_Nodes_Each_Level.csv", "w");
    if(fptr == NULL){
        printf("[Error] OpenFile : LD_Nodes_Each_Level.csv\n");
        exit(1);
    }
    #endif

    int loopCounter = 0;
    do{
        *hostAgainFlag          = 0;
        hipMemset(cudaAgainFlag, 0, sizeof(int));

        #ifdef RECORD_NodeProcessedEachLevel
        *hostNodeProcessedNum   = 0;
        hipMemset(cudaNodeProcessedNum, 0, sizeof(int));
        #endif
        LowDegreeColoring<<<_grid, _block>>>(_cudaColorArr, _cudaDAG_E, _cudaNodeInfo, cudaAgainFlag, _cudaNodeBitmap, _deviceCsrInfo, cudaNodeProcessedNum);
        
        hipDeviceSynchronize();

        hipMemcpy(hostAgainFlag, cudaAgainFlag, sizeof(int), hipMemcpyDeviceToHost);

        loopCounter ++;

        #ifdef RECORD_NodeProcessedEachLevel
        hipMemcpy(hostNodeProcessedNum, cudaNodeProcessedNum, sizeof(int), hipMemcpyDeviceToHost);
        printf("\tLoop[%d].nodeProcessedNum = %d\n", loopCounter, *hostNodeProcessedNum);
        fprintf(fptr, "%d,%d\n", loopCounter, *hostNodeProcessedNum);
        #endif

    }while(*hostAgainFlag == 1);

    printf("\tLoopCounter = %d\n", loopCounter);
    hipDeviceSynchronize();

    #ifdef RECORD_NodeProcessedEachLevel
    fclose(fptr);
    #endif

    printf("[Finish][ECL_GC_LowDegree_Vertex_Coloring]~\n\n");
}

#pragma endregion //ECL_GC_Low_Degree_Node_Coloring

#ifdef _NoUse_
__global__ void ECL_GC_InitSub( int _nodeID,            int* _cudaCsrV,     int* _cudaCsrE,
                                int* _cudaCsrDegree,    int* _cudaDAG_E,    int* _subKernelFlag,
                                int* _cudaDAG_nodeEndOffset, struct NodeInfo* _cudaDAG_E_Offset)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // degree 18
    // tid    0~31 只有 0~17有用，tid >= 18的都可以滾，只要tid < 18的做事就好
    if(tid < _cudaCsrDegree[_nodeID]){


        int neighborIndex   = tid;
        int neighborID      = _cudaCsrE[_cudaCsrV[_nodeID] + neighborIndex];
        
        int nodeDegree      = _cudaCsrDegree[_nodeID];
        int neighborDegree  = _cudaCsrDegree[neighborID];
        printf("node[%d] = {neighborIndex = %d, neighborID = %d, nodeDegree = %d, neighborDegree = %d}\n", _nodeID, neighborIndex, neighborID, nodeDegree, neighborDegree);
        
        if(tid == 0){
            //設置node在DAG_E的起始offset
            _cudaDAG_E_Offset[_nodeID].nodeStartOffset  = _cudaCsrV[_nodeID];
            //設置nodeEndOffset成起始的offset，且它之後會一直atomicAdd
            _cudaDAG_nodeEndOffset[_nodeID]             = _cudaCsrV[_nodeID]; 
        }
        // printf("cudaDAG_E_Offset[%d] = {startOffset = %d, EndOffset = %d}\n", _nodeID, _cudaDAG_E_Offset[_nodeID].nodeStartOffset, _cudaDAG_E_Offset[_nodeID].nodeEndOffset);
        //比較優先權(比較nodeDegree跟neighborDegree)
        if(nodeDegree < neighborDegree){
            //atomicAdd有問題，在這裡有+，但是不會去改到原本記憶體的值
            int nowOffset           = atomicAdd(&(_cudaDAG_nodeEndOffset[_nodeID]), 1);
            _cudaDAG_E[nowOffset]   = neighborID;
            printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", _nodeID, nowOffset, neighborID);
        }
        else if(nodeDegree == neighborDegree){ //break tie by nodeID and neighborID
            // if nodeID > neighborID，則代表neighbor會先塗，故neighbor優先於node，所以node的DAG要記錄neighbor
            if(_nodeID > neighborID){
            //atomicAdd有問題，在這裡有+，但是不會去改到原本記憶體的值
                int nowOffset           = atomicAdd(&(_cudaDAG_nodeEndOffset[_nodeID]), 1);
                _cudaDAG_E[nowOffset]   = neighborID;
                printf("node[%d] = {nowOffset = %d, neighbor = %d}\n", _nodeID, nowOffset, neighborID);
            }
            // else if nodeID < neighborID，則代表node會先塗，故node優先於neighbor，所以不用記錄東西
        }
    }

    if(tid == 0){
        printf("============node[%d] Finish\n", _nodeID);
        // *_subKernelFlag = 1;
    }
}
#endif